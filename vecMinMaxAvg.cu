#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>

#define N 8
__global__ void minimum(int *arr,int *minVal)
{
    int tid = threadIdx.x;
    
    *minVal = 999;
    atomicMin(minVal,arr[tid]);
}

__global__ void maximum(int *arr, int *maxVal)
{
    int tid = threadIdx.x;
    
    *maxVal  = 0;
    atomicMax(maxVal,arr[tid]);
}

__global__ void add(int *arr,int *avg) //only sums all the elements of the array
{
    int tid = threadIdx.x;
    *avg=0;
    atomicAdd(avg,arr[tid]);
}

int main(void)
{
    srand(time(NULL));
    int i;
    int arr[N],minVal,maxVal,addVal;
    
    int *dev_arr,*dev_min,*dev_max,*dev_add;
    
    hipMalloc((void **)&dev_arr, N*sizeof(int));
    hipMalloc((void **)&dev_min, N*sizeof(int));
    hipMalloc((void **)&dev_max, N*sizeof(int));
    hipMalloc((void **)&dev_add, N*sizeof(int));
    
    for(i=0;i<N;i++)
    {
        arr[i] = rand()%20 + 1;
    }
    
    printf("The array is : ");
   
    for(i=0;i<N;i++)
    {
        printf(" %d ",arr[i]);
    }
    
    hipMemcpy(dev_arr,arr,sizeof(arr),hipMemcpyHostToDevice);
    
    minimum<<<1,N>>>(dev_arr,dev_min);
    
    hipMemcpy(&minVal,dev_min,sizeof(minVal),hipMemcpyDeviceToHost);
    
    printf("The minimum value is : %d",minVal);
    
    
    
    hipMemcpy(dev_arr,arr,sizeof(arr),hipMemcpyHostToDevice);
    
    maximum<<<1,N>>>(dev_arr,dev_max);
    
    hipMemcpy(&maxVal,dev_max,sizeof(maxVal),hipMemcpyDeviceToHost);
    
    printf("The maximum value is : %d",maxVal);
    
    
    hipMemcpy(dev_arr,arr,sizeof(arr),hipMemcpyHostToDevice);
    
    add<<<1,N>>>(dev_arr,dev_add);
    
    hipMemcpy(&addVal,dev_add,sizeof(addVal),hipMemcpyDeviceToHost);
    
    printf("The average value is : %d",addVal/N);
    
    return 0;
}
