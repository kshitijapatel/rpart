#include <hip/hip_runtime.h>   //Max MIN
#include <stdio.h>
#include <time.h>
#define tbp 512
#define nblocks 1
__global__ void kernel_min(int *a, int *d)
{
    __shared__ int sdata[tbp]; //"static" shared memory

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = a[i];

    __syncthreads();
    for(unsigned int s=tbp/2 ; s >= 1 ; s=s/2)
    {
    if(tid < s)
    {
    if(sdata[tid] >sdata[tid + s])
    {
    sdata[tid] = sdata[tid + s];
    }
    }
    __syncthreads();
    }
    if(tid == 0 ) 
    {
    d[blockIdx.x] = sdata[0];
    }
}
int main()
{
    int i;
    const int N=tbp*nblocks;
    srand(time(NULL));

    int *a;
    a = (int*)malloc(N * sizeof(int));
    int *d;
    d = (int*)malloc(nblocks * sizeof(int));

    int *dev_a, *dev_d;

    hipMalloc((void **) &dev_a, N*sizeof(int));
    hipMalloc((void **) &dev_d, nblocks*sizeof(int));
    int mmm=100;
    for( i = 0 ; i < N ; i++)
    {
    a[i] = rand()% 100 + 5;
    //printf("%d ",a[i]);
    if(mmm>a[i]) mmm=a[i];

    }
    printf("");
    printf("");
    printf("");
    printf("");
    hipMemcpy(dev_a , a, N*sizeof(int),hipMemcpyHostToDevice);

    kernel_min <<<nblocks,tbp>>>(dev_a,dev_d);
    hipMemcpy(d, dev_d, nblocks*sizeof(int),hipMemcpyDeviceToHost);
    printf("cpu min %d, gpu_min = %d",mmm,d[0]);
    hipFree(dev_a);
    hipFree(dev_d);


    return 0;
}
